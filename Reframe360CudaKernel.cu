#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019-2021  Ronan LE MEILLAT, Stefan SIETZEN, Sylvain GRAVEL
* License Apache Software License 2.0
*/
#include ""
#include "hip/hip_vector_types.h"

#define M_PI	   3.14159265358979323846
#define M_PI_2     1.57079632679489661923   // pi/2
#define M_PI_4     0.785398163397448309616  // pi/4
#define M_1_PI     0.318309886183790671538  // 1/pi
#define M_2_PI     0.636619772367581343076  // 2/pi
#define OVERLAP 64
#define CUT 688
#define BASESIZE 4096 //OVERLAP and CUT are based on this size
#define TRUE	1
#define FALSE	0

enum Faces {
	TOP_LEFT,
	TOP_MIDDLE,
	TOP_RIGHT,
	BOTTOM_LEFT,
	BOTTOM_MIDDLE,
	BOTTOM_RIGHT,
	NB_FACES,
};

enum Direction {
	RIGHT,
	LEFT,
	UP,
	DOWN,
	FRONT,
	BACK,
	NB_DIRECTIONS,
};

enum Rotation {
	ROT_0,
	ROT_90,
	ROT_180,
	ROT_270,
	NB_ROTATIONS,
};

enum INPUT_FORMAT {
	EQUIRECTANGULAR,
	GOPRO_MAX,
	EQUIANGULAR_CUBEMAP,
	NB_INPUT_FORMAT,
};

__device__ float2 rotate_cube_face(float2 uv, int rotation)
{
	float2 ret_uv;

	switch (rotation) {
	case ROT_0:
		ret_uv = uv;
		break;
	case ROT_90:
		ret_uv.x = -uv.y;
		ret_uv.y = uv.x;
		break;
	case ROT_180:
		ret_uv.x = -uv.x;
		ret_uv.y = -uv.y;
		break;
	case ROT_270:
		ret_uv.x = uv.y;
		ret_uv.y = -uv.x;
		break;
	}
	return ret_uv;
}

__device__ float3 equirect_to_xyz(int2 xy, int2 size)
{
	float3 xyz;
	float phi = ((2.f * ((float)xy.x) + 0.5f) / ((float)size.x) - 1.f) * M_PI;
	float theta = ((2.f * ((float)xy.y) + 0.5f) / ((float)size.y) - 1.f) * M_PI_2;

	xyz.x = cos(theta) * sin(phi);
	xyz.y = sin(theta);
	xyz.z = cos(theta) * cos(phi);

	return xyz;
}

__device__ float2 xyz_to_cube(float3 xyz, int* direction, int* face)
{
	float phi = atan2(xyz.x, xyz.z);
	float theta = asin(xyz.y);
	float phi_norm, theta_threshold;
	int face_rotation;
	float2 uv;
	//int direction;

	if (phi >= -M_PI_4 && phi < M_PI_4) {
		*direction = FRONT;
		phi_norm = phi;
	}
	else if (phi >= -(M_PI_2 + M_PI_4) && phi < -M_PI_4) {
		*direction = LEFT;
		phi_norm = phi + M_PI_2;
	}
	else if (phi >= M_PI_4 && phi < M_PI_2 + M_PI_4) {
		*direction = RIGHT;
		phi_norm = phi - M_PI_2;
	}
	else {
		*direction = BACK;
		phi_norm = phi + ((phi > 0.f) ? -M_PI : M_PI);
	}

	theta_threshold = atan(cos(phi_norm));
	if (theta > theta_threshold) {
		*direction = DOWN;
	}
	else if (theta < -theta_threshold) {
		*direction = UP;
	}

	theta_threshold = atan(cos(phi_norm));
	if (theta > theta_threshold) {
		*direction = DOWN;
	}
	else if (theta < -theta_threshold) {
		*direction = UP;
	}

	switch (*direction) {
	case RIGHT:
		uv.x = -xyz.z / xyz.x;
		uv.y = xyz.y / xyz.x;
		*face = TOP_RIGHT;
		face_rotation = ROT_0;
		break;
	case LEFT:
		uv.x = -xyz.z / xyz.x;
		uv.y = -xyz.y / xyz.x;
		*face = TOP_LEFT;
		face_rotation = ROT_0;
		break;
	case UP:
		uv.x = -xyz.x / xyz.y;
		uv.y = -xyz.z / xyz.y;
		*face = BOTTOM_RIGHT;
		face_rotation = ROT_270;
		uv = rotate_cube_face(uv, face_rotation);
		break;
	case DOWN:
		uv.x = xyz.x / xyz.y;
		uv.y = -xyz.z / xyz.y;
		*face = BOTTOM_LEFT;
		face_rotation = ROT_270;
		uv = rotate_cube_face(uv, face_rotation);
		break;
	case FRONT:
		uv.x = xyz.x / xyz.z;
		uv.y = xyz.y / xyz.z;
		*face = TOP_MIDDLE;
		face_rotation = ROT_0;
		break;
	case BACK:
		uv.x = xyz.x / xyz.z;
		uv.y = -xyz.y / xyz.z;
		*face = BOTTOM_MIDDLE;
		face_rotation = ROT_90;
		uv = rotate_cube_face(uv, face_rotation);
		break;
	}

	return uv;
}

__device__ float2 xyz_to_eac(float3 xyz, int2 size)
{
	float pixel_pad = 2;
	float u_pad = pixel_pad / size.x;
	float v_pad = pixel_pad / size.y;

	int direction, face;
	int u_face, v_face;
	float2 uv = xyz_to_cube(xyz, &direction, &face);

	u_face = face % 3;
	v_face = face / 3;
	//eac expansion
	uv.x = M_2_PI * atan(uv.x) + 0.5f;
	uv.y = M_2_PI * atan(uv.y) + 0.5f;

	uv.x = (uv.x + u_face) * (1.f - 2.f * u_pad) / 3.f + u_pad;
	uv.y = uv.y * (0.5f - 2.f * v_pad) + v_pad + 0.5f * v_face;

	uv.x *= size.x;
	uv.y *= size.y;

	return uv;
}


__device__ int2 transpose_gopromax_overlap(int2 xy, int2 dim)
{
	int2 ret;
	int cut = dim.x * CUT / BASESIZE;
	int overlap = dim.x * OVERLAP / BASESIZE;
	if (xy.x < cut)
	{
		ret = xy;
	}
	else if ((xy.x >= cut) && (xy.x < (dim.x - cut)))
	{
		ret.x = xy.x + overlap;
		ret.y = xy.y;
	}
	else
	{
		ret.x = xy.x + 2 * overlap;
		ret.y = xy.y;
	}
	return ret;
}

__device__ float3 matMul(const float3 r012, const float3 r345, const float3 r678, float3 v){
	float3 outvec = { 0, 0, 0 };
	outvec.x = r012.x * v.x + r012.y * v.y + r012.z * v.z;
	outvec.y = r345.x * v.x + r345.y * v.y + r345.z * v.z;
	outvec.z = r678.x * v.x + r678.y * v.y + r678.z * v.z;
	return outvec;
}

__device__ float2 repairUv(float2 uv){
	float2 outuv;

	if(uv.x<0) {
		outuv.x = 1.0 + uv.x;
		}else if(uv.x > 1.0){
			outuv.x = uv.x -1.0;
		} else {
			outuv.x = uv.x;
		}

		if(uv.y<0) {
			outuv.y = 1.0 + uv.y;
		} else if(uv.y > 1.0){
			outuv.y = uv.y -1.0;
		} else {
			outuv.y = uv.y;
		}

	outuv.x = min(max(outuv.x, 0.0), 1.0);
	outuv.y = min(max(outuv.y, 0.0), 1.0);

	return outuv;
}

__device__ float2 polarCoord(float3 dir) {	
	float3 ndir = normalize(dir);
	float longi = -atan2(ndir.z, ndir.x);
	
	float lat = acos(-ndir.y);
	
	float2 uv;
	uv.x = longi;
	uv.y = lat;
	
	float2 M_PItwo = {M_PI, M_PI};
	uv /= M_PItwo;
	uv.x /= 2.0;
	float2 ones = {1.0, 1.0};
	uv = fmodf(uv, ones);
	return uv;
}


__device__ float3 fisheyeDir(float3 dir, const float3 r012, const float3 r345, const float3 r678) {

	if (dir.x == 0 && dir.y == 0)
		return matMul(r012, r345, r678, dir);
		
	dir.x = dir.x / dir.z;
	dir.y = dir.y / dir.z;
	dir.z = 1;
	
	float2 uv;
	uv.x = dir.x;
	uv.y = dir.y;
	float r = sqrtf(uv.x*uv.x + uv.y*uv.y);
	
	float phi = atan2f(uv.y, uv.x);
	
	float theta = r;
	
	float3 fedir = { 0, 0, 0 };
	fedir.x = sin(theta) * cos(phi);
	fedir.y = sin(theta) * sin(phi);
	fedir.z = cos(theta);

	fedir = matMul(r012, r345, r678, fedir);
	
	return fedir;
}

__device__ float3 tinyPlanetSph(float3 uv) {
	if (uv.x == 0 && uv.y == 0)
		return uv;

    float3 sph;
	float2 uvxy;
	uvxy.x = uv.x/uv.z;
	uvxy.y = uv.y/uv.z;

	float u  =length(uvxy);
	float alpha = atan2(2.0f, u);
	float phi = M_PI - 2*alpha;
	float z = cos(phi);
	float x = sin(phi);
	
	uvxy = normalize(uvxy);
	
	sph.z = z;
	
	float2 sphxy = uvxy * x;

	sph.x = sphxy.x;
	sph.y = sphxy.y;
	
	return sph;
}

__device__ float4 linInterpCol(float2 uv, const float* input, int width, int height){
	float4 outCol = {0,0,0,0};
	float i = floorf(uv.x);
	float j = floorf(uv.y);
	float a = uv.x-i;
	float b = uv.y-j;
	int x = (int)i;
	int y = (int)j;
	int x1 = (x < width - 1 ? x + 1 : x);
	int y1 = (y < height - 1 ? y + 1 : y);
	const int indexX1Y1 = ((y * width) + x) * 4;
	const int indexX2Y1 = ((y * width) + x1) * 4;
	const int indexX1Y2 = (((y1) * width) + x) * 4;
	const int indexX2Y2 = (((y1) * width) + x1) * 4;
	const int maxIndex = (width * height -1) * 4;
	
	if(indexX2Y2 < maxIndex){
		outCol.x = (1.0 - a)*(1.0 - b)*input[indexX1Y1] + a*(1.0 - b)*input[indexX2Y1] + (1.0 - a)*b*input[indexX1Y2] + a*b*input[indexX2Y2];
		outCol.y = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 1] + a*(1.0 - b)*input[indexX2Y1 + 1] + (1.0 - a)*b*input[indexX1Y2 + 1] + a*b*input[indexX2Y2 + 1];
		outCol.z = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 2] + a*(1.0 - b)*input[indexX2Y1 + 2] + (1.0 - a)*b*input[indexX1Y2 + 2] + a*b*input[indexX2Y2 + 2];
		outCol.w = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 3] + a*(1.0 - b)*input[indexX2Y1 + 3] + (1.0 - a)*b*input[indexX1Y2 + 3] + a*b*input[indexX2Y2 + 3];
	} else {
		outCol.x = input[indexX1Y1];
		outCol.y = input[indexX1Y1+ 1];
		outCol.z = input[indexX1Y1+ 2];
		outCol.w = input[indexX1Y1+ 3];
	}
	return outCol;
}

__device__ float2 get_original_coordinates(const float2 equirect_coordinates, int2 size, int transpose)
{
	int2 loc = { (int)equirect_coordinates.x, (int)equirect_coordinates.y };
	int2 eac_size = { size.x - 2 * (size.x * OVERLAP / BASESIZE),size.y };
	float3 xyz = equirect_to_xyz(loc, size);
	float2 uv = xyz_to_eac(xyz, eac_size);
	int2 xy;
	xy.x = roundf(uv.x);
	xy.y = roundf(uv.y);
	if (transpose>0)
	{
		xy = transpose_gopromax_overlap(xy, eac_size);
	}
	xy.y = size.y - (xy.y + 1);
	float2 ret = { (float)xy.x, (float)xy.y } ;
	return ret;
}

__device__ float2 get_original_gopromax_coordinates(const float2 equirect_coordinates, int2 size)
{
	return get_original_coordinates(equirect_coordinates, size, TRUE);
}

__global__ void Reframe360CudaKernel(int p_InputFormat, int p_Width, int p_Height, float* p_Fov, float* p_Tinyplanet, float* p_Rectilinear,
								const float* p_Input, float* p_Output, const float* r, int samples, bool bilinear)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
		const int index = ((y * p_Width) + x) * 4;

		float4 accum_col = {0, 0, 0, 0};

        float2 uv = { (float)x / p_Width, (float)y / p_Height };
        float aspect = (float)p_Width / (float)p_Height;
           
		for(int i=0; i<samples; i++){
			float fov = p_Fov[i];

		   float3 dir = { 0, 0, 0 };
		   dir.x = (uv.x * 2) - 1;
		   dir.y = (uv.y * 2) - 1;
		   dir.y /= aspect;
		   dir.z = fov;

		   float3 tinyplanet = tinyPlanetSph(dir);
		   tinyplanet = normalize(tinyplanet);

		   const float3 r012 = {r[i*9+0], r[i*9+1], r[i*9+2]};
		   const float3 r345 = {r[i*9+3], r[i*9+4], r[i*9+5]};
		   const float3 r678 = {r[i*9+6], r[i*9+7], r[i*9+8]};

		   tinyplanet = matMul(r012, r345, r678, tinyplanet);
		   float3 rectdir = matMul(r012, r345, r678, dir);

		   rectdir = normalize(rectdir);
		   dir = lerp(fisheyeDir(dir, r012, r345, r678), tinyplanet, p_Tinyplanet[i]);
		   dir = lerp(dir, rectdir, p_Rectilinear[i]);

		   float2 iuv = polarCoord(dir);
		   iuv = repairUv(iuv);

		   iuv.x *= (p_Width - 1);
		   iuv.y *= (p_Height - 1);
		   //get original coordinates

		   switch (p_InputFormat) {
		   case GOPRO_MAX:
			   iuv = get_original_gopromax_coordinates(iuv, { p_Width, p_Height });
			   break;
		   case EQUIANGULAR_CUBEMAP:
			   iuv = get_original_coordinates(iuv, { p_Width, p_Height }, FALSE);
			   break;
		   case EQUIRECTANGULAR:
			   break;
		   }

		   int x_new = iuv.x;
		   int y_new = iuv.y;

		   if ((x_new < p_Width) && (y_new < p_Height))
		   {
			   const int index_new = ((y_new * p_Width) + x_new) * 4;
			   
			   float4 interpCol;

			   if (bilinear){
				   interpCol = linInterpCol(iuv, p_Input, p_Width, p_Height);
			   }
			   else
			   {
				   interpCol = { p_Input[index_new + 0], p_Input[index_new + 1], p_Input[index_new + 2], p_Input[index_new + 3] };
			   }

			   accum_col.x += interpCol.x;
			   accum_col.y += interpCol.y;
			   accum_col.z += interpCol.z;
			   accum_col.w += interpCol.w;

			}
		}
		p_Output[index + 0] = accum_col.x / samples;
		p_Output[index + 1] = accum_col.y / samples;
		p_Output[index + 2] = accum_col.z / samples;
		p_Output[index + 3] = accum_col.w / samples;
   }
}


void RunCudaKernel(void* p_Stream, int p_inputFormat, int p_Width, int p_Height, float* p_Fov, float* p_Tinyplanet, float* p_Rectilinear, const float* p_Input, float* p_Output, const float* p_RotMat, int p_Samples, bool p_Bilinear)
{
	
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);
	hipStream_t stream = static_cast<hipStream_t>(p_Stream);

	float* dev_rmat;
	hipMalloc((void**)&dev_rmat, sizeof(float)*9*p_Samples);
	hipMemcpy((void*)dev_rmat, (void*)p_RotMat, sizeof(float)*9*p_Samples, hipMemcpyHostToDevice);

	float* dev_fov;
	hipMalloc((void**)&dev_fov, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_fov, (void*)p_Fov, sizeof(float)*p_Samples, hipMemcpyHostToDevice);

	float* dev_tinyplanet;
	hipMalloc((void**)&dev_tinyplanet, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_tinyplanet, (void*)p_Tinyplanet, sizeof(float)*p_Samples, hipMemcpyHostToDevice);
	
	float* dev_rectilinear;
	hipMalloc((void**)&dev_rectilinear, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_rectilinear, (void*)p_Rectilinear, sizeof(float)*p_Samples, hipMemcpyHostToDevice);


    Reframe360CudaKernel <<<blocks, threads, 0, stream >>> (p_inputFormat, p_Width, p_Height, dev_fov, dev_tinyplanet, dev_rectilinear,p_Input, p_Output, dev_rmat, p_Samples, p_Bilinear);


	hipFree( dev_rmat );
	hipFree( dev_fov );
	hipFree( dev_tinyplanet );
	hipFree( dev_rectilinear );

}
